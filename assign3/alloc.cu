#include "alloc.h"

void alloc(double* h_A, double* h_B, double* h_C, double* d_A, double* d_B, double* d_C, int A_no, int B_no, int C_no) {
	//Dynamic allocation of matrices with malloc() on CPU-side
	hipHostMalloc((void **)&h_A, A_no);
	hipHostMalloc((void **)&h_B, B_no);
	hipHostMalloc((void **)&h_C, C_no);
	//Allocation in global GPU memory
	hipMalloc((void **)&d_A, A_no);
	hipMalloc((void **)&d_B, B_no);
	hipMalloc((void **)&d_C, C_no);
}

void init(double* h_A, int A_no, double* h_B, int B_no, double* h_C, int C_no) {
	//Initialize matrix entries to one
	int i;
	for(i=0; i<A_no; i++) {
		h_A[i] = 1;	
	}
	for(i=0; i<B_no; i++) {
		h_B[i] = 1;
	}
	for(i=0; i<C_no; i++) {
		h_C[i] = 0;
	}
}

void transferToDevice(double* h_A, double* h_B, double* h_C, double* d_A, double* d_B, double* d_C, int A_no, int B_no, int C_no) {
	hipMemcpy(d_A, h_A, A_no, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, B_no, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, C_no, hipMemcpyHostToDevice);
}

void transferToHost(double* h_A, double* h_B, double* h_C, double* d_A, double* d_B, double* d_C, int A_no, int B_no, int C_no) {
	hipMemcpy(h_A, d_A, A_no, hipMemcpyDeviceToHost);
	hipMemcpy(h_B, d_B, B_no, hipMemcpyDeviceToHost);
	hipMemcpy(h_C, d_C, C_no, hipMemcpyDeviceToHost);
}

void freeall(double* h_A, double* h_B, double* h_C, double* d_A, double* d_B, double* d_C) {
	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

