int main(int argc, char **argv)
{
 // Allocate memory space on host and device
 h_data = malloc(...);
 hipMalloc(...);

 // Transfer data from host to device
 hipMemcpy(...);

 // Kernel launch
 kernel<<<Grid, Block>>>(...);
 hipDeviceSynchronize();

 // Transfer results from device to host
 hipMemcpy(...);

 // Free memory
 free(h_data);
 hipFree(...);
} 
