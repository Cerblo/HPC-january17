#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include "alloc.h"

extern "C" {
__global__ void kernel_gpu4(int m, int n, int k,double *A, double *B, double *C){

// 2D thread indices defining row and col of element
 	int j = blockIdx.x * blockDim.x + threadIdx.x;
 	int i = blockIdx.y * blockDim.y + threadIdx.y; //i goes from 1 out of 2 regarding lines of the matrix
	
	int l;

	if ((2*i)<m && (2*j)<n){
		C[2*i*n + 2*j]=0;
		C[(2*i+1)*n + 2*j]=0;
		C[2*i*n + 2*j+1]=0;
		C[(2*i+1)*n + 2*j+1]=0;

		for(l=0;l<k;l++){

			// 2 elements computed at the same time
			double a1=A[(2*i)*k +l];
			double a2=A[(2*i+1)*k + l];
			double b1=B[l*n+2*j];
			double b2=B[l*n+2*j+1];
			
			C[2*i*n + 2*j] += a1*b1;
			C[(2*i+1)*n + 2*j] += a2*b1;
			C[2*i*n + 2*j+1] += a1*b2;
			C[(2*i+1)*n + 2*j+1] += a2*b2;

		}
	}
	 
}

void matmult_gpu4(int m, int n, int k, double* A, double* B, double* C){


/*Declaring matrices (as arrays)	
	  matrix A: m, k
	  matrix B: k, n
	  matrix C: m, n		 */

	//Number of entries or matrix size
	int A_no = m*k*sizeof(double);
	int B_no = k*n*sizeof(double);
	int C_no = m*n*sizeof(double);
	
	double* d_A; 
	double* d_B; 
	double* d_C;

	int i; int j;

	/*for (i=0;i<m;i++){
	for (j=0;j<n;j++){
	printf("%f ", C[i*n + j]);
	}printf("\n");}printf("\n");*/

	//Allocation of memory for matrices
	alloc(&d_A, &d_B, &d_C, A_no, B_no, C_no);

	//Initialize matrix entries to one
	//init(h_A, A_no, h_B, B_no, h_C, C_no);

	//Transfer matrix to device
	transferToDevice(A, B,C, d_A, d_B, d_C, A_no, B_no, C_no);

	 // Kernel launch
	 int K = 16; //Size of the block
	dim3 dimgrid(ceil((double) n/K), ceil((double) m/K));
	dim3 dimblock(K/2,K/2);

	//printf("%f %f\n", ceil((double) m/(2*K)),ceil((double) n/K));
 	 kernel_gpu4<<<dimgrid, dimblock>>>(m,n,k,d_A,d_B,d_C);
	checkCudaErrors(hipDeviceSynchronize());
	
	//Transfer results from device to host
	transferToHost(A, B, C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Device Synchronization (and Cuda Error Check)
//	checkCudaErrors(hipDeviceSynchronize());
	
	//Freeing allocated memory
	freeall(d_A, d_B, d_C);
}
}

