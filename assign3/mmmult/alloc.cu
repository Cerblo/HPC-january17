#include "alloc.h"

void alloc(double** d_A, double** d_B, double** d_C, int A_no, int B_no, int C_no) {
	//Allocation in global GPU memory
	hipMalloc((void **)d_A, A_no);
	hipMalloc((void **)d_B, B_no);
	hipMalloc((void **)d_C, C_no);
}

void transferToDevice(double* h_A, double* h_B, double* h_C, double* d_A, double* d_B, double* d_C, int A_no, int B_no, int C_no) {
	hipMemcpy(d_A, h_A, A_no, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, B_no, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, C_no, hipMemcpyHostToDevice);
}

void transferToHost(double* h_A, double* h_B, double* h_C, double* d_A, double* d_B, double* d_C, int A_no, int B_no, int C_no) {
	hipMemcpy(h_A, d_A, A_no, hipMemcpyDeviceToHost);
	hipMemcpy(h_B, d_B, B_no, hipMemcpyDeviceToHost);
	hipMemcpy(h_C, d_C, C_no, hipMemcpyDeviceToHost);
}

void freeall(double* d_A, double* d_B, double* d_C) {
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

