#include "hip/hip_runtime.h"
#include "matmult_gpu1.h" 

extern "C" {
#include <omp.h>
__global__ void kernel_gpu1(int m, int n, int k, double* A, double* B, double* C) {
	int i, j, l;

	
	

	for ( i = 0; i < m; i++ ) {
		for ( j = 0; j < n; j++ ) {
			C[i*n+j] = 0;
			for ( l = 0; l < k; l++) {
				C[i*n+j] += A[i*k+l]*B[l*n+j];
			}
		}
	}
}


void matmult_gpu1(int m, int n, int k, double* A, double* B, double* C) { 
	
	//Number of entries or matrix size
	int A_no = m*k*sizeof(double);
	int B_no = k*n*sizeof(double);
	int C_no = m*n*sizeof(double);
	int i;int j;
	/*Initializing matrices (as arrays)	
	  matrix A: m, k
	  matrix B: k, n
	  matrix C: m, n		 */
	double* d_A; double* d_B; double* d_C;

	/*for (i=0;i<m;i++){
	for (j=0;j<n;j++){
	printf("%f ", C[i*n + j]);
	}printf("\n");}printf("\n");*/
	//Allocation of memory for matrices
	alloc(&d_A, &d_B, &d_C, A_no, B_no, C_no);

	//Transfer matrix to device
	transferToDevice(A, B, C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Kernel function call
	double time = omp_get_wtime();
	
	
	kernel_gpu1<<<1, 1>>>(m, n, k, d_A, d_B, d_C);
	double elapsed = omp_get_wtime()-time;
	printf("%5.10f\n", elapsed);
	//Device Synchronization (and Cuda Error Check)
	//checkCudaErrors(hipDeviceSynchronize());
	hipDeviceSynchronize();
	
	/*for (i=0;i<m;i++){
	for (j=0;j<n;j++){
	printf("%f ", C[i*n + j]);
	}printf("\n");}printf("\n");*/
	//Transfer results from device to host
	transferToHost(A, B, C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Freeing allocated memory
	freeall(d_A, d_B, d_C);
}
}/* end extern */
