
#include "alloc.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime_api.h>

extern "C" {
#include <cblas.h>
#include <omp.h>
void matmult_lib(int m, int n, int k, double* A, double* B, double* C) {

	/*Declaring matrices (as arrays)	
	  matrix A: m, k
	  matrix B: k, n
	  matrix C: m, n		 */
	
	//Number of entries or matrix size
	int A_no = m*k*sizeof(double);
	int B_no = k*n*sizeof(double);
	int C_no = m*n*sizeof(double);


	//Allocation of memory for matrices
	//alloc(h_A, h_B, h_C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Initialize matrix entries to one
	//init(h_A, A_no, h_B, B_no, h_C, C_no);

	//Transfer matrix to device
	//transferToDevice(h_A, h_B, h_C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Kernel function call
	//kernel_gpu1<<<1, 1>>>(m, n, k, d_A, d_B, d_C);
	
	//Transfer results from device to host
	//transferToHost(h_A, h_B, h_C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Device Synchronization (and Cuda Error Check)
	//checkCudaErrors(hipDeviceSynchronize());

	//Freeing allocated memory
	
	int alpha = 1;
        int beta =0;
	double time = omp_get_wtime();
	
	
	cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, \
		      m, n, k, alpha, A, k, B, n, beta, C, n);

	double elapsed = omp_get_wtime()-time;
	//printf("%5.10f\n", elapsed);

}

}
