#include "hip/hip_runtime.h"
__global__ void kernel_gpu1(int m, int n, int k, double* A, double* B, double* C) {
	int i, j, l;
	for ( i = 0; i < m; i++ ) {
		for ( j = 0; j < n; j++ ) {
			for ( l = 0; l < k; l++) {
				C[i*j+i] += A[i*l+i]*B[l*j+l];
			}
		}
	}
};


void matmult_gpu1(int m, int n, int k) { 

	/*Declaring matrices (as arrays)	
	  matrix A: m, k
	  matrix B: k, n
	  matrix C: m, n		 */
	double *h_A, *h_B, *h_C; //host adress
	double *d_A, *d_B, *d_C; //device address
	//Number of entries or matrix size
	int A_no = m*k*sizeof(double);
	int B_no = k*n*sizeof(double);
	int C_no = m*n*sizeof(double);

	//Allocation of memory for matrices
	alloc(h_A, h_B, h_C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Initialize matrix entries to one
	init(h_A, A_no, h_B, B_no, h_C, C_no);

	//Transfer matrix to device
	transferToDevice(h_A, h_B, h_C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Kernel function call
	kernel_gpu1<<<1, 1>>>(m, n, k, d_A, d_B, d_C);
	
	//Transfer results from device to host
	transferToHost(h_A, h_B, h_C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Device Synchronization (and Cuda Error Check)
	checkCudaErrors(hipDeviceSynchronize());

	//Freeing allocated memory
	freeall(h_A, h_B, h_C);
}

