#include "hip/hip_runtime.h"
#include "matmult_gpu1.h" 

extern "C" {

__global__ void kernel_gpu1(int m, int n, int k, double* A, double* B, double* C) {
	int i, j, l;
	for ( i = 0; i < m; i++ ) {
		for ( j = 0; j < n; j++ ) {
			for ( l = 0; l < k; l++) {
				C[i*n+j] += A[i*k+l]*B[l*n+j];
			}
		}
	}
}


void matmult_gpu1(int m, int n, int k) { 

	/*Declaring matrices (as arrays)	
	  matrix A: m, k
	  matrix B: k, n
	  matrix C: m, n		 */
	
	//Number of entries or matrix size
	int A_no = m*k*sizeof(double);
	int B_no = k*n*sizeof(double);
	int C_no = m*n*sizeof(double);

	double h_A[A_no];
	double h_B[B_no];
	double h_C[C_no];
	double d_A[A_no];
	double d_B[B_no];
	double d_C[C_no];

	//Allocation of memory for matrices
	alloc(h_A, h_B, h_C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Initialize matrix entries to one
	init(h_A, A_no, h_B, B_no, h_C, C_no);

	//Transfer matrix to device
	transferToDevice(h_A, h_B, h_C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Kernel function call
	//kernel_gpu1<<<1, 1>>>(m, n, k, d_A, d_B, d_C);
	
	//Transfer results from device to host
	transferToHost(h_A, h_B, h_C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Device Synchronization (and Cuda Error Check)
	//checkCudaErrors(hipDeviceSynchronize());
	hipDeviceSynchronize();

	//Freeing allocated memory
	freeall(h_A, h_B, h_C, d_A, d_B, d_C);
}
} /* end extern */
