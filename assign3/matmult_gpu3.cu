#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include "alloc.h"

extern "C" {
__global__ void kernel_gpu3(int m, int n, int k,double *A, double *B, double *C){

// 2D thread indices defining row and col of element
 	int j = blockIdx.x * blockDim.x + threadIdx.x;
 	int i = 2*(blockIdx.y * blockDim.y + threadIdx.y); //i goes from 1 out of 2 regarding lines of the matrix
	
	int l;

	if (i<m && j<n){

		for(l=0;l<k;l++){

			// 2 elements computed at the same time
			double b=B[l*n+j];

			C[i*n + j] += A[i*k+l]*b;
			C[(i+1)*n + j] += A[(i+1)*k+l]*b;

		}
	}
	 
}

void matmult_gpu3(int m, int n, int k){


/*Declaring matrices (as arrays)	
	  matrix A: m, k
	  matrix B: k, n
	  matrix C: m, n		 */

	//Number of entries or matrix size
	int A_no = m*k*sizeof(double);
	int B_no = k*n*sizeof(double);
	int C_no = m*n*sizeof(double);
	
	double h_A[A_no];
	double h_B[B_no];
	double h_C[C_no];
	double d_A[A_no];
	double d_B[B_no];
	double d_C[C_no];

	//Allocation of memory for matrices
	alloc(h_A, h_B, h_C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Initialize matrix entries to one
	init(h_A, A_no, h_B, B_no, h_C, C_no);

	//Transfer matrix to device
	transferToDevice(h_A, h_B, h_C, d_A, d_B, d_C, A_no, B_no, C_no);

	 // Kernel launch
	 int K = 16; //Size of the block
 	 kernel_gpu3<<<dim3(m/(2*K),n/K), dim3(K,K)>>>(m,n,k,d_A,d_B,d_C);
	 checkCudaErrors(hipDeviceSynchronize());
	
	//Transfer results from device to host
	transferToHost(h_A, h_B, h_C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Device Synchronization (and Cuda Error Check)
	checkCudaErrors(hipDeviceSynchronize());
	
	//Freeing allocated memory
	freeall(h_A, h_B, h_C,d_A, d_B, d_C);
}
}

