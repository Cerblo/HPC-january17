#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include "alloc.h"

extern "C" {
__global__ void kernel_gpu3(int m, int n, int k,double *A, double *B, double *C){

// 2D thread indices defining row and col of element
 	int j = blockIdx.x * blockDim.x + threadIdx.x;
 	int i = blockIdx.y * blockDim.y + threadIdx.y; //i goes from 1 out of 2 regarding lines of the matrix
	
	int l;

	if ((2*i)<m && j<n){
		C[2*i*n + j]=0;
		C[(2*i+1)*n + j]=0;

		for(l=0;l<k;l++){

			// 2 elements computed at the same time
			double b=B[l*n+j];
			
			C[2*i*n + j] += A[(2*i)*k+l]*b;
			C[(2*i+1)*n + j] += A[(2*i+1)*k+l]*b;

		}
	}
	 
}

void matmult_gpu3(int m, int n, int k, double* A, double* B, double* C){


/*Declaring matrices (as arrays)	
	  matrix A: m, k
	  matrix B: k, n
	  matrix C: m, n		 */

	//Number of entries or matrix size
	int A_no = m*k*sizeof(double);
	int B_no = k*n*sizeof(double);
	int C_no = m*n*sizeof(double);
	
	double* d_A; 
	double* d_B; 
	double* d_C;

	int i; int j;

	/*for (i=0;i<m;i++){
	for (j=0;j<n;j++){
	printf("%f ", C[i*n + j]);
	}printf("\n");}printf("\n");*/

	//Allocation of memory for matrices
	alloc(&d_A, &d_B, &d_C, A_no, B_no, C_no);

	//Initialize matrix entries to one
	//init(h_A, A_no, h_B, B_no, h_C, C_no);

	//Transfer matrix to device
	transferToDevice(A, B,C, d_A, d_B, d_C, A_no, B_no, C_no);

	 // Kernel launch
	 int K = 16; //Size of the block
	dim3 dimgrid(ceil((double) n/K), ceil((double) m/K));
	dim3 dimblock(K,K/2);

	//printf("%f %f\n", ceil((double) m/(2*K)),ceil((double) n/K));
 	 kernel_gpu3<<<dimgrid, dimblock>>>(m,n,k,d_A,d_B,d_C);
	hipDeviceSynchronize();
	
	//Transfer results from device to host
	transferToHost(A, B, C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Device Synchronization (and Cuda Error Check)
//	checkCudaErrors(hipDeviceSynchronize());
	
	//Freeing allocated memory
	freeall(d_A, d_B, d_C);
}
}

