#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include "alloc.h"

extern "C" {

__global__ void kernel_gpu2(int m, int n, int k, double *A, double *B, double *C){

// 2D thread indices defining row and col of element
 	int j = blockIdx.x * blockDim.x + threadIdx.x;
 	int i = blockIdx.y * blockDim.y + threadIdx.y; 
	
	int l;
	
	
	if (i<m && j<n){
		C[i*n + j] = 0;
		for(l=0;l<k;l++){
			
			C[i*n+j] += A[i*k+l]*B[l*n+j];

		}
	}
	 
}

void matmult_gpu2(int m, int n, int k, double* A, double* B, double* C){


/*Declaring matrices (as arrays)	
	  matrix A: m, k
	  matrix B: k, n
	  matrix C: m, n		 */
	
	//Number of entries or matrix size
	int A_no = m*k*sizeof(double);
	int B_no = k*n*sizeof(double);
	int C_no = m*n*sizeof(double);
	int i;int j;

	double* d_A; 
	double* d_B; 
	double* d_C;
	

	//Allocation of memory for matrices
	alloc(&d_A, &d_B, &d_C, A_no, B_no, C_no);

	//Initialize matrix entries to one
	//init(h_A, A_no, h_B, B_no, h_C, C_no);

	//Transfer matrix to device
	transferToDevice(A, B, C, d_A, d_B, d_C, A_no, B_no, C_no);

	 // Kernel launch
	 int K = 16; //Size of the block
	dim3 dimgrid(ceil((double) m/K), ceil((double) n/K));
	dim3 dimblock(K,K);
 	 kernel_gpu2<<<dimgrid, dimblock>>>(m,n,k,d_A,d_B,d_C);
	hipDeviceSynchronize();
	
	//Transfer results from device to host

	transferToHost(A, B, C, d_A, d_B, d_C, A_no, B_no, C_no);
	

	//Device Synchronization (and Cuda Error Check)
	//checkCudaErrors(hipDeviceSynchronize());
	
	/*for (i=0;i<m;i++){
	for (j=0;j<n;j++){
	printf("%f ", C[i*n + j]);
	}printf("\n");}printf("\n");*/
	//Freeing allocated memory
	freeall(d_A, d_B, d_C);
}
}

