#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include "alloc.h"

#define BLOCK_SIZE 16

extern "C" {

__global__ void kernel_gpu5(int m, int n, int k, double *A, double *B, double *C) {
	
	// Block rows and columns
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;

	// Thread rows and columns within Block
    	int row = threadIdx.y;
    	int col = threadIdx.x;

	double cbuffer = 0;
	
	__shared__ double a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double b[BLOCK_SIZE][BLOCK_SIZE];

	// Loop over sub-matrices A and B that are required to compute block
	int l_matrices = k/BLOCK_SIZE; 	// number of sub-matrices along
	int l; 			 	// iteration variable
	for (l = 0; l<l_matrices; l++) {
		
		// Load sub-matrices A and B into shared memory
		a[row][col] = A[row*k + col + l*BLOCK_SIZE + k*BLOCK_SIZE*blockCol];
		b[row][col] = B[row*n + col + l*BLOCK_SIZE*n + blockRow*n*BLOCK_SIZE];
		// Synchronize to make sure the sub-matrices are loaded
        	__syncthreads();

		// Multiply submatrices A and B together
		int count;
		for (count = 0; count<BLOCK_SIZE; count++) {
			cbuffer += a[row][count]*b[count][col];
		}
		// Sync before moving on to new submatrices
        	__syncthreads();
	}
	// each thread writes its result to matrix
	// alternatively, 
	C[row*n + col + blockRow*n + blockCol] = cbuffer; 
}


void matmult_gpu5(int m, int n, int k, double* A, double* B, double* C){

/*Declaring matrices (as arrays)	
	  matrix A: m, k
	  matrix B: k, n
	  matrix C: m, n		 */
	
	//Number of entries or matrix size
	int A_no = m*k*sizeof(double);
	int B_no = k*n*sizeof(double);
	int C_no = m*n*sizeof(double);

	double* d_A; 
	double* d_B; 
	double* d_C;
	

	//Allocation of memory for matrices
	alloc(&d_A, &d_B, &d_C, A_no, B_no, C_no);

	//Initialize matrix entries to one
	//init(h_A, A_no, h_B, B_no, h_C, C_no);

	//Transfer matrix to device
	transferToDevice(A, B, C, d_A, d_B, d_C, A_no, B_no, C_no);

	// Kernel launch
	int K = BLOCK_SIZE; //Size of the block
	dim3 dimgrid(ceil((double) m/K), ceil((double) n/K));
	dim3 dimblock(K,K);
 	kernel_gpu5<<<dimgrid, dimblock>>>(m,n,k,d_A,d_B,d_C);
	
	//Device Synchronization (and Cuda Error Check)
	//checkCudaErrors(hipDeviceSynchronize());
	hipDeviceSynchronize();
	
	//Transfer results from device to host
	transferToHost(A, B, C, d_A, d_B, d_C, A_no, B_no, C_no);
	
	/*int i, j;
	for (i=0;i<m;i++){
	for (j=0;j<n;j++){
	printf("%f ", C[i*n + j]);
	}printf("\n");}printf("\n");*/

	//Freeing allocated memory
	freeall(d_A, d_B, d_C);
}
}

