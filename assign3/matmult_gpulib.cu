
#include "alloc.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime_api.h>

#include "hipblas.h"

extern "C" {


void matmult_gpulib(int m, int n, int k, double* A, double* B, double* C) {

	int A_no = m*k*sizeof(double);
	int B_no = k*n*sizeof(double);
	int C_no = m*n*sizeof(double);
	

	double* d_A; 
	double* d_B; 
	double* d_C;
	int i; int j;

	/*for (i=0;i<m;i++){
	for (j=0;j<n;j++){
	printf("%f ", C[i*n + j]);
	}printf("\n");}printf("\n");*/

	//Allocation of memory for matrices

	alloc(&d_A, &d_B, &d_C, A_no, B_no, C_no);

	//Initialize matrix entries to one
	//init(h_A, A_no, h_B, B_no, h_C, C_no);

	//Transfer matrix to device 	
	transferToDevice(A, B, C, d_A, d_B, d_C, A_no, B_no, C_no);

 	//kernel_gpulib<<<dimgrid, dimblock>>>(m,n,k,d_A,d_B,d_C);
	
	

	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Column major -> leading dimensions of A B C change

	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, d_B, n, d_A, k, beta, d_C,n);
	
	//checkCudaErrors(hipDeviceSynchronize());
	//Transfer results from device to host
	transferToHost(A, B, C, d_A, d_B, d_C, A_no, B_no, C_no);

	//Device Synchronization (and Cuda Error Check)
//	checkCudaErrors(hipDeviceSynchronize());
	
	//Freeing allocated memory
	freeall(d_A, d_B, d_C);
	/*Declaring matrices (as arrays)	
	  matrix A: m, k
	  matrix B: k, n
	  matrix C: m, n		 */
	
	
	

}

}
