
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>


__host__ void 
mat_swap(double **A, double **B) {
  double *temp = *A;
  *A = *B;
  *B  = temp;
}

__global__ void 
jacobian(double *OLD, double *NEW, double *f, int size, int max_it, \
              double h) {

  /* initializing iteration variables */
  int i,j;

    for (i = 1; i < size - 1; i++) {
      for (j = 1; j < size - 1; j++) {
        NEW[i * size + j] = 0.25 * ( OLD[(i-1) * size + j] + OLD[(i+1) * size + j] + OLD[i * size + (j-1)]\
                          + OLD[i * size + (j+1)] + h * h * f[i * size + j]);
//printf("%5.2f ", NEW[i * size + j]);
      }
//printf("\n");
    }
}
