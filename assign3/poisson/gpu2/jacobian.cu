
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>


__host__ void 
mat_swap(double **A, double **B) {
  double *temp = *A;
  *A = *B;
  *B  = temp;
}

__global__ void 
jacobian(double *OLD, double *NEW, double *f, int size, int max_it, \
              double h) {

  /* initializing iteration variables */
  int i,j;
// 1 is added to row and column because the boundaries are not to be changed
int row = blockIdx.x * blockDim.x + threadIdx.x + 1;
int col = blockIdx.y * blockDim.y + threadIdx.y + 1;

if (row < size - 1 && col < size - 1) {
        NEW[row * size + col] = 0.25 * ( OLD[(row - 1) * size + col] + OLD[(row + 1) * size + col] + OLD[row * size + (col-1)]\
                          + OLD[row * size + (col+1)] + h * h * f[row * size + col]);
    }
}
