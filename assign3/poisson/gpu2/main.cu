#include "hip/hip_runtime.h"
#include <stdio.h>
#include "datatools.h"
#include "init.h"
#include "jacobian.h"
#include <hip/hip_runtime_api.h>
#include <math.h>

int main(int argc, char **argv) {

  int size, N, max_it, total_size, k;
  double h, guess;
  double *h_Uold, *h_Unew, *h_f;
  double *d_Uold, *d_Unew, *d_f;

  if (argc < 3) {
  	printf("Wrong input\n");
  	return 0;
  }

  N = atoi(argv[1]);
  guess = 15;

  max_it = atoi(argv[2]);

  size = N + 2;
  h = 2.0 / (N + 1);
total_size = size * size * sizeof(double);

  checkCudaErrors(hipHostMalloc((void**)&h_Uold, total_size));
  checkCudaErrors(hipHostMalloc((void**)&h_Unew, total_size));
  checkCudaErrors(hipHostMalloc((void**)&h_f, total_size));

checkCudaErrors(hipMalloc((void**)&d_Uold, total_size));
checkCudaErrors(hipMalloc((void**)&d_Unew, total_size));
checkCudaErrors(hipMalloc((void**)&d_f, total_size));

  init_u(size, h_Uold, guess);
  init_u(size, h_Unew, guess);
  init_f(N, h_f);

checkCudaErrors(hipMemcpy(d_Uold, h_Uold, total_size, hipMemcpyHostToDevice));
checkCudaErrors(hipMemcpy(d_Unew, h_Unew, total_size, hipMemcpyHostToDevice)); 
checkCudaErrors(hipMemcpy(d_f, h_f, total_size, hipMemcpyHostToDevice)); 


dim3 threadsPerBlock(16, 16);
dim3 numBlocks(ceil(N/16.0), ceil(N/16.0));
k = 0;
while(k < max_it) {

mat_swap(&d_Uold, &d_Unew);

  jacobian<<<numBlocks, threadsPerBlock>>>(d_Uold, d_Unew, d_f, size, max_it, h);
checkCudaErrors(hipDeviceSynchronize());

k++;
}

checkCudaErrors(hipMemcpy(h_Unew, d_Unew, total_size, hipMemcpyDeviceToHost));

int i;
for (i = 0; i < size * size; i++) {
if (i % size == 0)
printf("\n");
printf("%5.1f ", h_Unew[i]);}
printf("\n");
//}


checkCudaErrors(hipHostFree(h_Uold));
checkCudaErrors(hipHostFree(h_Unew));
checkCudaErrors(hipHostFree(h_f));

checkCudaErrors(hipFree(d_Uold));
checkCudaErrors(hipFree(d_Unew));
checkCudaErrors(hipFree(d_f));

  return 0;
}
