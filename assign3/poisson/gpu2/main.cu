#include "hip/hip_runtime.h"
#include <stdio.h>
#include "datatools.h"
#include "init.h"
#include "jacobian.h"
#include <math.h>

int main(int argc, char **argv) {

  int size, N, max_it, total_size, k;
  double h, guess;
  double *h_Uold, *h_Unew, *h_f;
  double *d_Uold, *d_Unew, *d_f;

  if (argc < 3) {
  	printf("Wrong input\n");
  	return 0;
  }

  N = atoi(argv[1]);
  guess = 15;

  max_it = atoi(argv[2]);

  size = N + 2;
  h = 2.0 / (N + 1);
total_size = size * size * sizeof(double);

  hipHostMalloc((void**)&h_Uold, total_size);
  hipHostMalloc((void**)&h_Unew, total_size);
  hipHostMalloc((void**)&h_f, total_size);

hipMalloc((void**)&d_Uold, total_size);
hipMalloc((void**)&d_Unew, total_size);
hipMalloc((void**)&d_f, total_size);

  init_u(size, h_Uold, guess);
  init_u(size, h_Unew, guess);
  init_f(N, h_f);

hipMemcpy(d_Uold, h_Uold, total_size, hipMemcpyHostToDevice);
hipMemcpy(d_Unew, h_Unew, total_size, hipMemcpyHostToDevice); 
hipMemcpy(d_f, h_f, total_size, hipMemcpyHostToDevice); 


dim3 threadsPerBlock(16, 16);
dim3 numBlocks(ceil(N/16.0), ceil(N/16.0));
k = 0;
while(k < max_it) {

mat_swap(&d_Uold, &d_Unew);

  jacobian<<<numBlocks, threadsPerBlock>>>(d_Uold, d_Unew, d_f, size, max_it, h);
hipDeviceSynchronize();

k++;
}

hipMemcpy(h_Unew, d_Unew, total_size, hipMemcpyDeviceToHost);


hipHostFree(h_Uold);
hipHostFree(h_Unew);
hipHostFree(h_f);

hipFree(d_Uold);
hipFree(d_Unew);
hipFree(d_f);

  return 0;
}
