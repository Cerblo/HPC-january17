
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>


__host__ void 
mat_swap(double **A, double **B) {
  double *temp = *A;
  *A = *B;
  *B  = temp;
}

__global__ void 
jacobian_0(double *d0_OLD, double *d1_OLD, double *NEW, double *f, int size, int max_it, double h) {

int row = blockIdx.x * blockDim.x + threadIdx.x + 1;
int col = blockIdx.y * blockDim.y + threadIdx.y + 1;
if (row == size/2 - 1 && col < size - 1) {
        NEW[row * size + col] = 0.25 * ( d0_OLD[(row - 1) * size + col] + d1_OLD[0 * size + col] + d0_OLD[row * size + (col-1)]\
                          + d0_OLD[row * size + (col+1)] + h * h * f[row * size + col]);
}
if (row < size/2 - 1 && col < size - 1) {
        NEW[row * size + col] = 0.25 * ( d0_OLD[(row - 1) * size + col] + d0_OLD[(row + 1) * size + col] + d0_OLD[row * size + (col-1)]\
                          + d0_OLD[row * size + (col+1)] + h * h * f[row * size + col]);
    }
}

// Function taking care of the bottom of the matrix
__global__ void 
jacobian_1(double *d0_OLD, double *d1_OLD, double *NEW, double *f, int size, int max_it, \
              double h) {

  /* initializing iteration variables */
// This time row don't start with 1 because there is no boundaries (since it's the bottom
int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y + 1;

// Separate the case when communicating with the other GPU is necessary
if (row == 0 && col < size - 1) {
        NEW[row * size + col] = 0.25 * ( d0_OLD[(size / 2 - 1)* size + col] + d1_OLD[(row + 1) * size + col] + d1_OLD[row * size + (col-1)]\
                          + d1_OLD[row * size + (col+1)] + h * h * f[row * size + col]);
}
else if (row < size/2 - 1 && col < size - 1) {
        NEW[row * size + col] = 0.25 * ( d1_OLD[(row - 1) * size + col] + d1_OLD[(row + 1) * size + col] + d1_OLD[row * size + (col-1)]\
                          + d1_OLD[row * size + (col+1)] + h * h * f[row * size + col]);
    }
}
