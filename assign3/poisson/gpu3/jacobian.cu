
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>


__host__ void 
mat_swap(double **A, double **B) {
  double *temp = *A;
  *A = *B;
  *B  = temp;
}

__global__ void 
jacobian_0(double *d0_OLD, double *d1_OLD, double *NEW, double *f, int size, int max_it, double h) {

int row = blockIdx.x * blockDim.x + threadIdx.x + 1;
int col = blockIdx.y * blockDim.y + threadIdx.y + 1;

if (row == size/2 - 1 && col < size - 1) {
printf("jac 0 input - %i,%i: %f\n", row, col, d1_OLD[0 * size + col]);
        NEW[row * size + col] = 0.25 * ( d0_OLD[(row - 1) * size + col] + d1_OLD[0 * size + col] + d0_OLD[row * size + (col-1)]\
                          + d0_OLD[row * size + (col+1)] + h * h * f[row * size + col]);
}
else if (row < size/2 - 1 && col < size - 1) {
printf("jac 0 input - %i,%i: %f\n", row, col, d0_OLD[row * size + col]);
        NEW[row * size + col] = 0.25 * ( d0_OLD[(row - 1) * size + col] + d0_OLD[(row + 1) * size + col] + d0_OLD[row * size + (col-1)]\
                          + d0_OLD[row * size + (col+1)] + h * h * f[row * size + col]);
//printf("jac 0 output- %i,%i: %f\n", row, col, NEW[row * size + col]);
    }
}


__global__ void 
jacobian_1(double *d0_OLD, double *d1_OLD, double *NEW, double *f, int size, int max_it, \
              double h) {

  /* initializing iteration variables */
// 1 is added to row and column because the boundaries are not to be changed
int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y + 1;

if (row == 0 && col < size - 1) {
printf("input jac 1 - %i,%i: %f\n", row + size/2, col, d0_OLD[(size / 2 - 2) * size + col]);
        NEW[row * size + col] = 0.25 * ( d0_OLD[(size / 2 - 1)* size + col] + d1_OLD[(row + 1) * size + col] + d1_OLD[row * size + (col-1)]\
                          + d1_OLD[row * size + (col+1)] + h * h * f[row * size + col]);
//printf("jac 1 - %i,%i: %f\n", row + size/2, col, NEW[row * size + col]);
}
else if (row < size/2 - 1 && col < size - 1) {
printf("input jac 1 - %i,%i: %f\n", row + size/2, col, d1_OLD[row * size + col]);
        NEW[row * size + col] = 0.25 * ( d1_OLD[(row - 1) * size + col] + d1_OLD[(row + 1) * size + col] + d1_OLD[row * size + (col-1)]\
                          + d1_OLD[row * size + (col+1)] + h * h * f[row * size + col]);
    }
}
