#include "hip/hip_runtime.h"
#include <stdio.h>
#include "datatools.h"
#include "init.h"
#include "jacobian.h"
//#include <hip/hip_runtime_api.h>
#include <math.h>

int main(int argc, char **argv) {

  int size, N, max_it, total_size, k;
  double h, guess;
  double *h_Uold, *h_Unew, *h_f;
  double *d0_Uold, *d0_Unew, *d0_f;
  double *d1_Uold, *d1_Unew, *d1_f;

  if (argc < 3) {
  	printf("Wrong input\n");
  	return 0;
  }

  N = atoi(argv[1]);
  guess = 15;

  max_it = atoi(argv[2]);

  size = N + 2;
  h = 2.0 / (N + 1);
total_size = size * size * sizeof(double);

  hipHostMalloc((void**)&h_Uold, total_size);
  hipHostMalloc((void**)&h_Unew, total_size);
  hipHostMalloc((void**)&h_f, total_size);

  init_u(size, h_Uold, guess);
  init_u(size, h_Unew, guess);
  init_f(N, h_f);

hipSetDevice(6);

hipMalloc((void**)&d0_Uold, total_size / 2);
hipMalloc((void**)&d0_Unew, total_size / 2);
hipMalloc((void**)&d0_f, total_size / 2);

hipMemcpy(d0_Uold, h_Uold, total_size / 2, hipMemcpyHostToDevice);
hipMemcpy(d0_Unew, h_Unew, total_size / 2, hipMemcpyHostToDevice); 
hipMemcpy(d0_f, h_f, total_size / 2, hipMemcpyHostToDevice); 

hipSetDevice(7);

hipMalloc((void**)&d1_Uold, total_size / 2);
hipMalloc((void**)&d1_Unew, total_size / 2 );
hipMalloc((void**)&d1_f, total_size / 2);

hipMemcpy(d1_Uold, h_Uold + size * size / 2, total_size / 2, hipMemcpyHostToDevice);
hipMemcpy(d1_Unew, h_Unew + size * size / 2, total_size / 2, hipMemcpyHostToDevice); 
hipMemcpy(d1_f, h_f + size * size / 2, total_size / 2, hipMemcpyHostToDevice); 

hipDeviceEnablePeerAccess(7, 6);

dim3 threadsPerBlock(16, 16);
dim3 numBlocks(ceil(N/16.0/2), ceil(N/16.0));
k = 0;
while(k < max_it) {

hipSetDevice(6);
mat_swap(&d0_Uold, &d0_Unew);
mat_swap(&d1_Uold, &d1_Unew);

  jacobian_0<<<numBlocks, threadsPerBlock>>>(d0_Uold, d1_Uold, d0_Unew, d0_f, size, max_it, h);
hipDeviceSynchronize();



hipSetDevice(7);
  jacobian_1<<<numBlocks, threadsPerBlock>>>(d0_Uold, d1_Uold, d1_Unew, d1_f, size, max_it, h);
hipDeviceSynchronize();

k++;
}

hipSetDevice(6);
hipMemcpy(h_Unew, d0_Unew, total_size / 2, hipMemcpyDeviceToHost);

hipSetDevice(7);
hipMemcpy(h_Unew + size * size / 2, d1_Unew, total_size / 2, hipMemcpyDeviceToHost);

int i;
for (i = 0; i < size * size; i++) {
if (i % size == 0)
printf("\n");
printf("%5.1f ", h_Unew[i]);}
printf("\n");


hipHostFree(h_Uold);
hipHostFree(h_Unew);
hipHostFree(h_f);

hipSetDevice(6);
hipFree(d0_Uold);
hipFree(d0_Unew);
hipFree(d0_f);


hipSetDevice(7);
hipFree(d1_Uold);
hipFree(d1_Unew);
hipFree(d1_f);
  return 0;
}
