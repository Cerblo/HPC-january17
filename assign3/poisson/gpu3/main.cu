#include "hip/hip_runtime.h"
#include <stdio.h>
#include "datatools.h"
#include "init.h"
#include "jacobian.h"
#include <math.h>

int main(int argc, char **argv) {

  int size, N, max_it, total_size, k;
  double h, guess;
  double *h_Uold, *h_Unew, *h_f;
  double *d0_Uold, *d0_Unew, *d0_f;
  double *d1_Uold, *d1_Unew, *d1_f;

  if (argc < 3) {
  	printf("Wrong input\n");
  	return 0;
  }

  N = atoi(argv[1]);
  guess = 15;

  max_it = atoi(argv[2]);

  size = N + 2;
  h = 2.0 / (N + 1);
total_size = size * size * sizeof(double);

  hipHostMalloc((void**)&h_Uold, total_size);
  hipHostMalloc((void**)&h_Unew, total_size);
  hipHostMalloc((void**)&h_f, total_size);

  init_u(size, h_Uold, guess);
  init_u(size, h_Unew, guess);
  init_f(N, h_f);

hipSetDevice(0);

hipMalloc((void**)&d0_Uold, total_size / 2);
hipMalloc((void**)&d0_Unew, total_size / 2);
hipMalloc((void**)&d0_f, total_size / 2);

hipMemcpy(d0_Uold, h_Uold, total_size / 2, hipMemcpyHostToDevice);
hipMemcpy(d0_Unew, h_Unew, total_size / 2, hipMemcpyHostToDevice); 
hipMemcpy(d0_f, h_f, total_size / 2, hipMemcpyHostToDevice); 

hipSetDevice(1);

hipMalloc((void**)&d1_Uold, total_size / 2);
hipMalloc((void**)&d1_Unew, total_size / 2 );
hipMalloc((void**)&d1_f, total_size / 2);

hipMemcpy(d1_Uold, h_Uold + size * size / 2, total_size / 2, hipMemcpyHostToDevice);
hipMemcpy(d1_Unew, h_Unew + size * size / 2, total_size / 2, hipMemcpyHostToDevice); 
hipMemcpy(d1_f, h_f + size * size / 2, total_size / 2, hipMemcpyHostToDevice); 

hipDeviceEnablePeerAccess(0, 1);

dim3 threadsPerBlock(16, 16);
dim3 numBlocks(ceil(N/16.0/2), ceil(N/16./20));
k = 0;
while(k < max_it) {

hipSetDevice(0);
mat_swap(&d0_Uold, &d0_Unew);
mat_swap(&d1_Uold, &d1_Unew);

  jacobian_0<<<numBlocks, threadsPerBlock>>>(d0_Uold, d1_Uold, d0_Unew, d0_f, size, max_it, h);
hipDeviceSynchronize();


hipSetDevice(1);
  jacobian_1<<<numBlocks, threadsPerBlock>>>(d0_Uold, d1_Uold, d1_Unew, d1_f, size, max_it, h);
hipDeviceSynchronize();

k++;
}

hipSetDevice(0);
hipMemcpy(h_Unew, d0_Unew, total_size / 2, hipMemcpyDeviceToHost);

hipSetDevice(1);
hipMemcpy(h_Unew + size * size / 2, d1_Unew, total_size / 2, hipMemcpyDeviceToHost);
hipHostFree(h_Uold);
hipHostFree(h_Unew);
hipHostFree(h_f);

hipSetDevice(0);
hipFree(d0_Uold);
hipFree(d0_Unew);
hipFree(d0_f);


hipSetDevice(1);
hipFree(d1_Uold);
hipFree(d1_Unew);
hipFree(d1_f);
  return 0;
}
